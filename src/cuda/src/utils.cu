#include <hip/hip_runtime.h>

#include <stdio.h>
#include <hipblas.h>
#include <cstring>



extern "C" void check_Cuda_Errors(hipError_t err, const char* msg, const char* file, int line) {
    if (err != hipSuccess) {
        printf("CUDA Error in %s at line %d\n", file, line);
        printf("%s - %s\n", msg, hipGetErrorString(err));
        exit(0);
    }
}


const char* cublas_Get_Error_String(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "CUBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:
            return "CUBLAS_STATUS_LICENSE_ERROR";
    }
    return "UNKNOWN CUBLAS ERROR";
}

extern "C" void check_Cublas_Errors(hipblasStatus_t status, const char* msg, const char* file, int line) {

    const char* err = cublas_Get_Error_String(status);

    if (strcmp(err, "CUBLAS_STATUS_SUCCESS") != 0) {
        printf("CUBLAS Error in %s at line %d\n", file, line);
        printf("%s - %s\n", msg, err);
        exit(0);
    }
}


